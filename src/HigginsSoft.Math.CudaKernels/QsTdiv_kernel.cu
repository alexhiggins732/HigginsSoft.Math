
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 /**
  * CUDA Kernel Device code
  *
  * Computes the vector addition of A and B into C. The 3 vectors have the same
  * number of elements numElements.
  */

extern "C" __global__ void
QsTDiv(int* n, int* bsmooth, int* gf2, int* div, const int* primes, int N, int P)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
	{
		int mask = 0;
		for (int j = 0; n[i] > 1 && j < P; j++)
		{
			mask = 1 << j;
			while (n[i] % primes[j] == 0)
			{
				n[i] /= primes[j];
				gf2[i] ^= mask;
				div[i] |= mask;
			}
		}
		bsmooth[i] = n[i] == 1 ? 1 : 0;
	}
}
