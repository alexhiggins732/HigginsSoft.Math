
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
float[][] data_h;
CudaDeviceVariable<CUdeviceptr> data_d;
CUdeviceptr[] ptrsToData_h; //represents data_d on host side
CudaDeviceVariable<float>[] arrayOfarray_d; //Array of CudaDeviceVariables to manage memory, source for pointers in ptrsToData_h.

int sizeX = 512;
int sizeY = 256;

data_h = new float[sizeX][];
arrayOfarray_d = new CudaDeviceVariable<float>[sizeX];
data_d = new CudaDeviceVariable<CUdeviceptr>(sizeX);
ptrsToData_h = new CUdeviceptr[sizeX];
for (int x = 0; x < sizeX; x++)
{
    data_h[x] = new float[sizeY];
    arrayOfarray_d[x] = new CudaDeviceVariable<float>(sizeY);
    ptrsToData_h[x] = arrayOfarray_d[x].DevicePointer;
    //ToDo: init data on host...
}
//Copy the pointers once:
data_d.CopyToDevice(ptrsToData_h);

//Copy data:
for (int x = 0; x < sizeX; x++)
{
    arrayOfarray_d[x].CopyToDevice(data_h[x]);
}

//Call a kernel:
kernel.Run(data_d.DevicePointer , other parameters);

//kernel in *cu file:
//__global__ void kernel(float** data_d, ...)
*/
//Correspondend kernel:
extern "C"
__global__ void KernelJagged(float* data, size_t pitch, int dimX, int dimY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= dimX || y >= dimY)
        return;

    //pointer arithmetic: add y*pitch to char* pointer as pitch is given in bytes,
    //which gives the start of line y. Convert to float* and add x, to get the
    //value at entry x of line y:
    float value = *(((float*)((char*)data + y * pitch)) + x);

    *(((float*)((char*)data + y * pitch)) + x) = value + 1;

    //Or simpler if you don't like pointers:
    float* line = (float*)((char*)data + y * pitch);
    float value2 = line[x];
}