
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

extern "C" __global__ void
vectorAdd(const unsigned long long*A, const unsigned long long*B, unsigned long long*C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for (int j = 0; j < 250000; j++)
    {
        if (i < numElements)
        {
            C[i] = A[i] + B[i];
        }
    }
}

