
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 /**
  * CUDA Kernel Device code
  *
  * Computes the vector addition of A and B into C. The 3 vectors have the same
  * number of elements numElements.
  */

/*
int dimX = 512;
int dimY = 512;
float[] array_host = new float[dimX * dimY];
CudaPitchedDeviceVariable<float> arrayPitched_d = new CudaPitchedDeviceVariable<float>(dimX, dimY);
for (int y = 0; y < dimY; y++)
{
    for (int x = 0; x < dimX; x++)
    {
        array_host[y * dimX + x] = x * y;
    }
}

arrayPitched_d.CopyToDevice(array_host);
kernel.Run(arrayPitched_d.DevicePointer, arrayPitched_d.Pitch, dimX, dimY);
*/
//Correspondend kernel:
extern "C"
__global__ void KernelPitched(float* data, size_t pitch, int dimX, int dimY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= dimX || y >= dimY)
        return;

    //pointer arithmetic: add y*pitch to char* pointer as pitch is given in bytes,
    //which gives the start of line y. Convert to float* and add x, to get the
    //value at entry x of line y:
    float value = *(((float*)((char*)data + y * pitch)) + x);

    *(((float*)((char*)data + y * pitch)) + x) = value + 1;

    //Or simpler if you don't like pointers:
    float* line = (float*)((char*)data + y * pitch);
    float value2 = line[x];
}